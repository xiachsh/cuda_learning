#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "common.cuh"

#define N 128*1024
#define MAX_OFFSET 128



__global__ void assignValue (float *a,float *b)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	a[idx] = 1.0f * idx;
	b[idx] = 2.0f * idx;
}


__global__ void memcpyOffset(float *a,float *b,int offset)
{
        int idx = blockDim.x * blockIdx.x + threadIdx.x + offset;

        a[idx] = b[idx];
}


int main()
{
	hipError_t err;
	hipEvent_t start,end;

	size_t bytes = sizeof(float) * (N + MAX_OFFSET) ;
	
	
	float * a,*b;
	float * c;
	
	c = (float *) malloc(bytes);
	CHECK_CUDA_ERR( hipMalloc(&a,bytes) );
	CHECK_CUDA_ERR( hipMalloc(&b,bytes) );

	CHECK_CUDA_ERR(	hipEventCreate(&start) );
	CHECK_CUDA_ERR( hipEventCreate(&end)   );
	

	assignValue<<<128,1024>>> (a,b);

	CHECK_LAST_CUDA_ERR

	int i = 0;
	float ms = 0;
	for (i = 0;i<MAX_OFFSET;i++ ) {
		
		CHECK_CUDA_ERR(	hipEventRecord(start,0));
		memcpyOffset<<<128,1024>>>(a,b,i);
		CHECK_LAST_CUDA_ERR
		CHECK_CUDA_ERR( hipEventRecord(end,0));
		CHECK_CUDA_ERR( hipDeviceSynchronize());

		CHECK_CUDA_ERR( hipEventElapsedTime(&ms,start,end));

		printf("offset :%d time :%f milli second\n",i,ms);
	}
	CHECK_CUDA_ERR( hipEventDestroy(start) );
	CHECK_CUDA_ERR( hipEventDestroy(end) );



	CHECK_CUDA_ERR( hipFree(a) );		
	CHECK_CUDA_ERR( hipFree(b) );		
	free(c);

	return 0;
}
