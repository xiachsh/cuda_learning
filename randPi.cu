
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <limits.h>
#include <hiprand/hiprand.h>



#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

__global__ void largerThanOne(float *x,float *y,unsigned int *pnts)
{
	int idx = threadIdx.x;
	if (x[idx] * x[idx] + y[idx] * y[idx] <= 1) {
		pnts[idx] = 1;
	}
}


int main(void)
{

	float * pntsX;
	float * pntsY;

	unsigned int *pnts_h = 0;
	unsigned int *pnts = 0;
	unsigned int totalPnts = 0;

	hiprandGenerator_t gen;
	
	int elems = 1024;
	int iteration = 1;
	int nBytes = elems * sizeof(float);	

	hipMalloc((void **) (&pntsX),nBytes);
	hipMalloc((void **) (&pntsY),nBytes);

	CURAND_CALL(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT));
	CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL));
	CURAND_CALL(hiprandGenerateUniform(gen,pntsX,elems));

	
	CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen,2345ULL));
	CURAND_CALL(hiprandGenerateUniform(gen,pntsY,elems));
	

	hipMalloc((void **)&pnts,sizeof(unsigned int)*elems);
	hipMemset(pnts,0,sizeof(unsigned int)*elems);
	
	largerThanOne<<<1,elems>>>(pntsX,pntsY,pnts);
        pnts_h = (unsigned int *) malloc(sizeof(unsigned int)*elems);	
	hipMemcpy(pnts_h,pnts,sizeof(unsigned int)*elems,hipMemcpyDeviceToHost);
	int i = 0;
	int _pnts = 0;
	for (i=0;i<elems;i++)
		_pnts += pnts_h[i];
	printf("pi is roughly about %f\n",(float)_pnts * 4 / elems);


	 CURAND_CALL(hiprandDestroyGenerator(gen));
	hipFree(pnts);
	hipFree(pntsX);
	hipFree(pntsY);

	free(pnts_h);

	return 0;
}
