#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERR(x) 				\
	if ( (err = x) != hipSuccess) 	{ 		\
	printf("%d failed with error :%s\n",__LINE__,hipGetErrorString(err));	\
	exit(1);							\
	}

#define CHECK_LAST_ERR 					\
	if ( (err = hipGetLastError()) != hipSuccess) 	{ 		\
	printf("last cuda call failed with error :%s\n",hipGetErrorString(err));	\
	exit(1);							\
	}

	



#define WIDTH 16
#define HEIGHT 16



__global__ void print_element(float * input, int pitch)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;

	float * ptr =  (float*)((char*)input + bid * pitch) + tid;


	printf("[%2d,%2d]:%10.2f\n",bid,tid,*ptr);


}

int main()
{
	hipError_t err;
	float * buf;
	float * dBuf;

	size_t pitch;
	size_t bytes = WIDTH * HEIGHT * sizeof(float);
	size_t bytesPerRow = WIDTH * sizeof(float);
	

	CHECK_CUDA_ERR ( hipHostMalloc(&buf,bytes, hipHostMallocDefault) );
	CHECK_CUDA_ERR ( hipMallocPitch(&dBuf,&pitch,bytesPerRow,HEIGHT) );

	int i  = 0;
	for (;i<HEIGHT;i++) {
		int j = 0;
		for (;j<WIDTH;j++) {
			buf[i * WIDTH + j] = i * WIDTH + j;	
		}	
	}

	CHECK_CUDA_ERR ( hipMemcpy2D(dBuf,pitch,buf,WIDTH * sizeof(float),WIDTH * sizeof(float),WIDTH,hipMemcpyDeviceToHost) );
	
	print_element<<<WIDTH,HEIGHT>>> (dBuf,pitch);

	

	
	CHECK_CUDA_ERR( hipHostFree(buf) );
	CHECK_CUDA_ERR( hipFree(dBuf) );
	return 0;
}
