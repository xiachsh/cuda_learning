#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



__global__ void kernel1()
{
	printf("kernel #1\n");
}

__global__ void kernel2()
{
	printf("kernel #2\n");
}
int main(int argc,char **argv)
{

	printf("Testing multiple kernel launch and show in-order execution of two kernels \n");

	int nThreadsPerBlock = 32;
	
	int blocks = 128;
	hipStream_t s1;
	hipStream_t s2;


	hipStreamCreate(&s1);
	hipStreamCreate(&s2);

	kernel1<<<blocks,nThreadsPerBlock,0,s1>>> ();	
	hipError_t err = hipGetLastError();               
        if (err != hipSuccess ) printf("cuda function failure at line %d :%s \n",__LINE__,hipGetErrorString(err));  
	kernel2<<<blocks,nThreadsPerBlock,1,s2>>> ();	
	err = hipGetLastError();               
        if (err != hipSuccess )  printf("cuda function failure at line %d :%s \n",__LINE__,hipGetErrorString(err));  
	
	hipDeviceSynchronize();
	return 0 ;
}

