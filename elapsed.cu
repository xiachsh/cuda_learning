#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#define N  1024 * 1024



#define CHECK_CUDA_ERR(x)   {			\
	err = x;				\
	if (err != hipSuccess) { 		\
		printf("cuda error with %s in line %d\n",hipGetErrorString(err),__LINE__);	\
		exit(1);									\
	} }

int main()
{
	
	hipError_t err;
	hipEvent_t start_event,stop_event;

	CHECK_CUDA_ERR ( hipEventCreate(&start_event) );
	CHECK_CUDA_ERR ( hipEventCreate(&stop_event)  );




	void * buf = malloc(sizeof(float) * N);

	void * dBuf;
	 CHECK_CUDA_ERR ( hipMalloc(&dBuf,sizeof(float) * N) );

	 CHECK_CUDA_ERR ( hipEventRecord(start_event,0) ) ;
	 CHECK_CUDA_ERR ( hipMemcpy(dBuf,buf,sizeof(float) * N, hipMemcpyHostToDevice));
	 CHECK_CUDA_ERR ( hipEventRecord(stop_event,0) );
	 CHECK_CUDA_ERR ( hipDeviceSynchronize() );
	float ms = 100.f;

	 CHECK_CUDA_ERR ( hipEventElapsedTime(&ms,start_event,stop_event));


	printf("%f m second cost\n",ms);


	CHECK_CUDA_ERR ( hipEventDestroy(start_event) );
	CHECK_CUDA_ERR ( hipEventDestroy(stop_event)  );
	free(buf);
	CHECK_CUDA_ERR ( hipFree(dBuf) );


	return 0;





}
